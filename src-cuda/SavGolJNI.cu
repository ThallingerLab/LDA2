#include "hip/hip_runtime.h"
#include <jni.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "SavGolJNI.h"

#define UPPER true
#define LOWER false
#define MIN_RADIUS 12

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))

struct gh_values // global host values
{
	float *d_values;
	float *h_values;
	float *precalcPow;
};

int blockSizePowers;
int blockSizeSmooth;

/* =============================================================================
* Checks if a CUDA capable device is present
*
* @param env:       JNI environment
* @param thisObj:   reference to "this" Java object
*
* @return:          true if there is a cuda capable device
*/
JNIEXPORT jboolean JNICALL Java_at_tugraz_genome_lda_quantification_SavGolJNI_cudaCapableDeviceNative
(JNIEnv *env, jobject thisObj)
{

	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
		return false;
	else
		return true;

}

/* =============================================================================
* Checks if an error accured on the device
*
* @param ans: a cuda function call
*/
// http://stackoverflow.com/a/14038590/6914637
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void
gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file,
			line);
		if (abort)
			exit(code);
	}
}

/* =============================================================================
* Finds the lowest or highest index within the range
*
* @param dtIndx:        The point to extend the range
* @param range:         The time which the index should be extendet to
* @param posDirection:  Gives the information to move in the array up or down
* @param values:        Contains time, raw data and smoothed data
* @param numberOfScans: Length of values
*
* @return the lowest or highest index
*/
int calcBoundIndex(int dtIndx, float range, bool posDirection, float *values,
	int numberOfScans)
{
	int boundIndex = dtIndx;
	if (posDirection == UPPER)
	{
		while (boundIndex < (numberOfScans - 1)
			&& (values[boundIndex * 4] - values[dtIndx * 4]) < range)
		{
			++boundIndex;
		}
	}
	else
	{
		while (boundIndex > 0
			&& (values[dtIndx * 4] - values[boundIndex * 4]) < range)
		{
			--boundIndex;
		}
	}
	return boundIndex;
}

/* =============================================================================
* Pre calculates the 4th root of the raw/smoothed values
*
* @param powers:        holds the result of the calculations
* @param values:        Contains time, raw data and smoothed data
* @param numberOfScans: Length of values
* @param copyDirection: location of the smoothed data
* 						    ( 0->d_values[][2], 1->d_values[][3] )
*/
__global__
void
precalculatePowers(float *powers, float *values, int numberOfScans,
	int copyDirection)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while (idx < numberOfScans)
	{
		if (values[4 * idx + 2 + copyDirection] > 1)
			powers[idx] = rsqrt(rsqrt(values[4 * idx + 2 + copyDirection]));
		else
			powers[idx] = 1.0f;
		idx += blockDim.x * gridDim.x;
	}
}

/* =============================================================================
* Finds the lowest or highest index within the range
*
* @param boundIndex:    The calculated lowest or highest index
* @param idx:           The point to extend the range
* @param posDirection:  Gives the information to move in the array up or down
* @param numberOfScans: Length of values
* @param values:        Contains time, raw data and smoothed data
* @param range:	        The time which the index should be extendet to
*
* @return the lowest or highest index
*/
__device__
void
deviceCalcBoundIndex(int *boundIndex, int idx, bool posDirection,
	int numberOfScans, float *values, float range)
{
	*boundIndex = idx;
	if (posDirection == UPPER)
	{
		while (*boundIndex < (numberOfScans - 1)
			&& (values[4 * (*boundIndex) + 0] - values[4 * idx + 0]) < range)
		{
			++*boundIndex;
		}
	}
	else
	{
		while (*boundIndex > 0
			&& (values[4 * idx + 0] - values[4 * (*boundIndex) + 0]) < range)
		{
			--*boundIndex;
		}
	}
}

/* =============================================================================
* performs LU decomposition
*
* @param mtrx
* @param order
* @param indx
* @param vv
*/
__device__
void
myLUDcmp(float mtrx[][5], int order, int *indx, float *vv)
{
	float big, dum, sum, temp;
	int i, j, k;
	int imax = 0;
	float d = 1.0f;

	for (i = 0; i < order; i++)
	{
		big = 0.0f;
		for (j = 0; j < order; j++)
			if ((temp = fabs(mtrx[i][j])) > big)
				big = temp;
		vv[i] = 1.0F / big;
	}

	for (j = 0; j < order; j++)
	{
		for (i = 0; i < j; i++)
		{
			sum = mtrx[i][j];
			for (k = 0; k < i; k++)
				sum -= mtrx[i][k] * mtrx[k][j];
			mtrx[i][j] = sum;
		}
		big = 0.0f;
		for (i = j; i < order; i++)
		{
			sum = mtrx[i][j];
			for (k = 0; k < j; k++)
				sum -= mtrx[i][k] * mtrx[k][j];
			mtrx[i][j] = sum;
			dum = vv[i] * fabs(sum);
			if (dum >= big)
			{
				big = dum;
				imax = i;
			}
		}
		if (j != imax)
		{
			for (k = 0; k < order; k++)
			{
				dum = mtrx[imax][k];
				mtrx[imax][k] = mtrx[j][k];
				mtrx[j][k] = dum;
			}
			d *= -1;
			vv[imax] = vv[j];
		}
		indx[j] = imax;
		if (mtrx[j][j] == 0.0)
			mtrx[j][j] = 1E-20f;
		if (j != order)
		{
			dum = 1.0F / mtrx[j][j];
			for (i = j + 1; i < order; i++)
				mtrx[i][j] *= dum;
		}
	}
}

/* =============================================================================
* performs LU backsubstition
*
* @param mtrx
* @param order
* @param indx
* @param vec
*/
__device__
void
myLUBksb(float mtrx[][5], int order, int *indx, float *vec)
{
	float sum;
	int ii, ip;
	int i, j;

	ii = -1;
	for (i = 0; i < order; i++)
	{
		ip = indx[i];
		sum = vec[ip];
		vec[ip] = vec[i];
		if (ii != -1)
			for (j = ii; j < i; j++)
				sum -= mtrx[i][j] * vec[j];
		else if (sum != 0)
			ii = i;
		vec[i] = sum;
	}
	for (i = order - 1; i >= 0; i--)
	{
		sum = vec[i];
		for (j = i + 1; j < order; j++)
			sum -= mtrx[i][j] * vec[j];
		vec[i] = sum / mtrx[i][i];
	}
}

/* =============================================================================
* Does a Savitzky Golay Filter around a given point
*
* @param val:           The smoothed value at idx
* @param g_values:      Stored on the global memory.
*                           Contains time, raw data and smoothed data
* @param g_powers:      Precalculated powers
* @param idx:           Index of point
* @param lower:         Lower range border
* @param upper:         Upper range border
* @param order:         Order of polynome
* @param copyDirection: Location of the smoothed data
*                           ( 0->g_values[][2], 1->g_values[][3] )
* @param numberOfScans: Length of g_values
* @param radius:        Maximum number of points within the range
* @param shared_memory: Dynamically allocated shared memory
* @param blockSize:     Block size
*/
__device__
void
SavGolFilter(float *val, float *g_values, float *g_powers, int idx, int lower,
	int upper, int order, int copyDirection, int numberOfScans,
	int radius, float *shared_memory, int blockSize)
{
	// -------------------------------------------------------------------------
	// copy values[] and powers[] from the global to the shared memory

	// setting new indices
	int s_idx = threadIdx.x + radius;
	int offset = blockSize * (idx / blockSize) - radius;
	lower = lower - offset;
	upper = upper - offset;

	// init shared variables
	float* values = &shared_memory[0];
	float* powers = &shared_memory[4 * blockSize + 2 * 4 * radius];

	// copy the data
	for (int i = 0; i < 4; i++)
		values[4 * s_idx + i] = g_values[4 * idx + i];
	powers[s_idx] = g_powers[idx];

	// copy the data outside of the radius
	if (threadIdx.x < radius)
	{
		if (idx - radius >= 0)
		{
			for (int i = 0; i < 4; i++)
			{
				values[4 * (s_idx - radius) + i] = g_values[4 * (idx - radius) + i];
			}
			powers[s_idx - radius] = g_powers[idx - radius];
		}
		if (idx + blockSize < numberOfScans)
		{
			for (int i = 0; i < 4; i++)
			{
				values[4 * (s_idx + blockSize) + i] = g_values[4 * (idx + blockSize) + i];
			}
			powers[s_idx + blockSize] = g_powers[idx + blockSize];
		}
	}

	// wait until every thread within a block is done with copying to shared
	// memory
	__syncthreads();
	
	float x = values[4 * s_idx + 0];
	float sum = 0.0f;
	float adding = 0.0f;
	float mtrx[5][5];
	float vec[5];
	int indx[5];
	float vv[4];

	// -------------------------------------------------------------------------
	// get "mtrx"

	float sums[9];

	// calculate the sums for mtrx
	for (int m = 0; m < 2 * order + 1; m++)
	{
		sums[m] = 0.0f;

		for (int k = lower; k <= upper; k++)
		{
			adding = 1.0f;

			for (int l = 0; l < m; l++)
			{
				adding = adding * (values[4 * k + 0] - x);
			}
			if (m == 0)
				adding = 1.0f;

			adding = adding * powers[k];
			sums[m] += adding;

		}
	}

	// assign the sums to the mtrx
	for (int i = 0; i <= order; i++)
	{
		for (int j = i; j <= order; j++)
		{
			mtrx[i][j] = sums[i + j];
			mtrx[j][i] = sums[i + j];
		}
	}

	// -------------------------------------------------------------------------
	// LU decomposition

	myLUDcmp(mtrx, order + 1, indx, vv);

	// -------------------------------------------------------------------------
	// get "vec"

	for (int i = 0; i <= order; i++)
	{
		sum = 0.0f;
		for (int k = lower; k <= upper; k++)
		{
			sum += powf(values[4 * k + 0] - x, i)
				* values[4 * k + 2 + copyDirection] * powers[k];
		}
		vec[i] = sum;
	}

	// -------------------------------------------------------------------------
	// LU backsubstition

	myLUBksb(mtrx, order + 1, indx, vec);

	*val = vec[0];
}

extern __shared__ float shared_memory[];
/* =============================================================================
* smoothes raw spectrum at a specific point.
*
* @param range:         number of seconds around given points
* @param threshold:     minimum value to pass back
* @param powers:        precalculated powers
* @param values:        Contains time, raw data and smoothed data
* @param numberOfScans: Length of g_values
* @param copyDirection: location of the smoothed data
*                           ( 0->g_values[][2], 1->g_values[][3] )
* @param radius:        Maximum number of points within the range
* @param blockSize:     Block size
* @param startOffset:   Difference between precalculation start and smooth start
* @param stopOffset:    Difference between precalculation stop and smooth stop
*/
__global__
void
SmoothDataPoint(float range, float threshold, float *powers, float *values,
	int numberOfScans, int copyDirection, int radius,
	int blockSize, int startOffset, int stopOffset)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int order = 4;
	float val = 0;

	while (idx < numberOfScans)
	{
		// ---------------------------------------------------------------------
		// get boundary of indices which should be included into the calculation

		int lower;
		deviceCalcBoundIndex(&lower, idx, LOWER, numberOfScans, values, range);
		int upper;
		deviceCalcBoundIndex(&upper, idx, UPPER, numberOfScans, values, range);

		while (upper - lower < 10)
		{
			if (lower > 0)
				--lower;
			if (upper < numberOfScans - 1)
				++upper;
			if (lower <= 0 && upper >= numberOfScans - 1)
				break;
		}

		// ---------------------------------------------------------------------
		// Get the smoothed value

		order = order < upper - lower - 1 ? order : upper - lower - 1;

		if (order < 1)
		{
			val = values[4 * idx + 2 + copyDirection];
		}
		else
		{
			SavGolFilter(&val, values, powers, idx, lower, upper, order,
				copyDirection, numberOfScans, radius, shared_memory,
				blockSize);
			if (val < threshold)
			{
				val = threshold;
			}
		}
		
		if (idx < startOffset || idx >= (numberOfScans - stopOffset))
		{
			val = values[4 * idx + 2 + copyDirection];
		}

		values[4 * idx + 3 - copyDirection] = val;

		idx += blockDim.x * gridDim.x;
	}
}

/* =============================================================================
* Smooths the raw spectrum
*
* @param values:            Contains time, raw data and smoothed data
* @param numberOfScans:     Length of g_values
* @param range:             number of seconds around given points
* @param repeats:           how often the spectrum should be smoothed
* @param startSmoothScan:   start index of the spectrum
* @param stopSmoothScan:    end index of the spectrum
* @param address:           Adress of values struct for each thread
*/
void prepareSmooth(float *values, int numberOfScans,
	float range, int repeats, int startSmoothScan,
	int stopSmoothScan, long address)
{
	struct gh_values *ptr = (struct gh_values *)address;
	// -------------------------------------------------------------------------
	// Calculate the minimum intensity of the raw data and
	// store it into threshold:

	float threshold = values[1];
	for (int i = 0; i < numberOfScans; i++)
	{
		if (values[i * 4 + 1] < threshold)
		{
			threshold = values[i * 4 + 1];
		}
	}

	// -------------------------------------------------------------------------
	// Set the start and stop point of the smoothing range

	int startScan = 0;
	if (startSmoothScan > -1)
		startScan = startSmoothScan;
	int preCalcStart =
		calcBoundIndex(startScan, range, LOWER, values, numberOfScans) - 10 ;
	if (preCalcStart < 0)
		preCalcStart = 0;

	int stopScan = numberOfScans;
	if (stopSmoothScan > -1)
		stopScan = stopSmoothScan;
	int preCalcStop =
		calcBoundIndex(stopScan, range, UPPER, values, numberOfScans) + 10;
	if (preCalcStop > numberOfScans)
		preCalcStop = numberOfScans;

	// -------------------------------------------------------------------------
	// Adjust the length of values

	int smooth_length = preCalcStop - preCalcStart;
	float *smooth_values = values + preCalcStart * 4;

	// -------------------------------------------------------------------------
	// Setting the size of the block and grid dimension

	int gridSizePowers = (smooth_length + blockSizePowers - 1) / blockSizePowers;
	int gridSizeSmooth = (smooth_length + blockSizeSmooth - 1) / blockSizeSmooth;

	// -------------------------------------------------------------------------
	// Copy the data from the host to the device

	gpuErrchk(
		hipMemcpy(ptr->d_values, smooth_values, 4 * smooth_length * sizeof(float),
			hipMemcpyHostToDevice));

	// -------------------------------------------------------------------------
	// Calculate the shared memory allocation size
    
	float min_time_diff = smooth_values[(smooth_length - 1) * 4] - smooth_values[0];
	for (int i = 0; i < smooth_length - 1; i++)
	{
		float time_diff = smooth_values[(i + 1) * 4] - smooth_values[i * 4];
		min_time_diff = MIN(min_time_diff, time_diff);
	}
	int radius = MAX((range / min_time_diff + 1) * 2, MIN_RADIUS);

	int shared_space_values = 4 * blockSizeSmooth + 2 * 4 * radius;
	int shared_space_powers = blockSizeSmooth + 2 * radius;
	int shared_space = shared_space_values + shared_space_powers;

	// -------------------------------------------------------------------------
	// Smooth the raw data

	for (int i_rep = 0; i_rep < repeats; i_rep++)
	{
		precalculatePowers <<<gridSizePowers, blockSizePowers >>>(ptr->precalcPow,
			ptr->d_values, smooth_length, i_rep % 2);
		SmoothDataPoint <<<gridSizeSmooth, blockSizeSmooth,
			shared_space * sizeof(float) >>>(range, threshold, ptr->precalcPow,
			ptr->d_values, smooth_length, i_rep % 2, radius, blockSizeSmooth,
			startScan-preCalcStart, preCalcStop-stopScan);
	}
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	// -------------------------------------------------------------------------
	// copying back the data from the device

	gpuErrchk(
		hipMemcpy(smooth_values, ptr->d_values, 4 * smooth_length * sizeof(float),
			hipMemcpyDeviceToHost));

	if (repeats % 2 == 1)
	{
		for (int i = 2; i < 4 * smooth_length; i += 4)
		{
			values[i + preCalcStart * 4] = smooth_values[i + 1];
		}
	}
	else
	{
		for (int i = 2; i < 4 * smooth_length; i += 4)
		{
			values[i + preCalcStart * 4] = smooth_values[i];
		}
	}
}

/* =============================================================================
* Allocates memory on the graphicscard
*
* @param env:           JNI environment
* @param thisObj:       reference to "this" Java object
* @param mallocSize:    the size of allocated memory
*/
JNIEXPORT jlong JNICALL Java_at_tugraz_genome_lda_quantification_SavGolJNI_initMallocNative(JNIEnv *env,
	jobject thisObj,
	jint mallocSize)
{
	struct gh_values *ptr = (struct gh_values *)malloc(sizeof(*ptr));

	ptr->h_values = (float *)malloc(4 * (int)mallocSize * sizeof(float));
	gpuErrchk(hipMalloc(&ptr->d_values, 4 * (int)mallocSize * sizeof(float)));
	gpuErrchk(hipMalloc(&ptr->precalcPow, (int)mallocSize * sizeof(float)));

	int minGridSize;
	gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSizePowers,
		precalculatePowers, 0, 0));
	gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSizeSmooth,
		SmoothDataPoint, 0, 0));

	return (jlong)ptr;
}

/* =============================================================================
* Receives variables from the Java class SavGolJNI
*
* @param env:               JNI environment
* @param thisObj:           reference to "this" Java object
* @param j_values:          Contains time, raw data and smoothed data
* @param numberOfScans:	    Length of g_values
* @param range:             number of seconds around given points
* @param repeats:           how often the spectrum should be smoothed
* @param startSmoothScan:   start index of the spectrum
* @param stopSmoothScan:    end index of the spectrum
* @param address:           address of the device variables
*
*/
JNIEXPORT jfloatArray JNICALL Java_at_tugraz_genome_lda_quantification_SavGolJNI_SmoothNative(JNIEnv *env,
	jobject thisObj,
	jobjectArray j_value, jint numberOfScans, jfloat range, jint repeats,
	jint startSmoothScan, jint stopSmoothScan, jlong address)
{ 
	struct gh_values *ptr = (struct gh_values *)address;
	// -------------------------------------------------------------------------
	// Copies the values from Java to C

	// input variables
	int i, j;
	int len2 = 4;
	float* values;
	values = ptr->h_values; 

	// output variables
	jfloatArray j_smoothed = env->NewFloatArray(numberOfScans);
	jfloat *smoothed = env->GetFloatArrayElements(j_smoothed, NULL);

	// copying the values
	for (i = 0; i<numberOfScans; ++i)
	{
		jfloatArray oneDim = (jfloatArray)env->GetObjectArrayElement(j_value, i);
		jfloat *element = env->GetFloatArrayElements(oneDim, 0);

		for (j = 0; j<len2; ++j)
		{
			values[i*len2 + j] = element[j];
		}

		env->ReleaseFloatArrayElements(oneDim, element, JNI_ABORT);
		env->DeleteLocalRef(oneDim);
	}

	// -------------------------------------------------------------------------
	// Smoothing of the raw data

	prepareSmooth(values, (int)numberOfScans, (float)range, (int)repeats,
		(int)startSmoothScan, (int)stopSmoothScan, (long)address);
		
	// -------------------------------------------------------------------------
	// Copy data back to Java

	// copying data
	for (i = 0; i < numberOfScans; i++)
	{
		smoothed[i] = values[i*len2 + 2];
	}

	// release the smoothed array
	env->ReleaseFloatArrayElements(j_smoothed, (jfloat *)smoothed, 0);
	//free(values);

	// returns the smoothed values
	return j_smoothed;
}

/* =============================================================================
* Frees the Allocated space on the graphics card
*
* @param env:       JNI environment
* @param thisObj:   reference to "this" Java object
* @param address:   address of the device variables
*/
JNIEXPORT void JNICALL Java_at_tugraz_genome_lda_quantification_SavGolJNI_FreesNative(JNIEnv *env, jobject thisObj,
	jlong address)
{
	struct gh_values *ptr = (struct gh_values *)address;

	free(ptr->h_values);
	gpuErrchk(hipFree(ptr->d_values));
	gpuErrchk(hipFree(ptr->precalcPow));

	free(ptr);

	return;
}
